#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"

using OperatorClass = cutlass::arch::OpClassSimt;
using ArchTag = cutlass::arch::Sm86;
using ThreadblockShape = cutlass::gemm::device::DefaultGemmConfiguration<
	OperatorClass, ArchTag, float, float, float, float>::ThreadblockShape;
/// Warp-level tile size (concept: GemmShape)
using WarpShape = cutlass::gemm::device::DefaultGemmConfiguration<
	OperatorClass, ArchTag, float, float, float, float>::WarpShape;
/// Instruction-level tile size (concept: GemmShape)
using InstructionShape = cutlass::gemm::device::DefaultGemmConfiguration<
	OperatorClass, ArchTag, float, float, float, float>::InstructionShape;
/// Epilogue output operator
using EpilogueOutputOp = cutlass::gemm::device::DefaultGemmConfiguration<
	OperatorClass, ArchTag, float, float, float, float>::EpilogueOutputOp;
using ThreadblockSwizzle = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;
static const int kStages = cutlass::gemm::device::DefaultGemmConfiguration<OperatorClass, ArchTag, float, float, float, float>::kStages;
/// Access granularity of A matrix in units of elements
static const int kAlignmentA = cutlass::gemm::device::DefaultGemmConfiguration<OperatorClass, ArchTag, float, float, float, float>::kAlignmentA;
/// Access granularity of B matrix in units of elements
static const int kAlignmentB = cutlass::gemm::device::DefaultGemmConfiguration<OperatorClass, ArchTag, float, float, float, float>::kAlignmentB;
static const bool kSplitKSerial = false;
template<typename LayoutA, typename LayoutB, typename LayoutC>
using CutlassGemmKernel = typename cutlass::gemm::kernel::DefaultGemm<
  float,
  LayoutA,
  kAlignmentA,
  float,
  LayoutB,
  kAlignmentB,
  float,
  LayoutC,
  float,
  OperatorClass,
  ArchTag,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOutputOp,
  ThreadblockSwizzle,
  kStages,
  kSplitKSerial,
  cutlass::gemm::device::DefaultGemmConfiguration<OperatorClass, ArchTag, float, float, float, float>::Operator
>::GemmKernel;

template<typename T>
void check(T result, char const *const func, const char *const file, int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
				file, line, static_cast<unsigned int>(result), func);
		// Make sure we call CUDA Device Reset before exiting
		abort();
	}
}
#define checkCudaErrors(val)           check((val), #val, __FILE__, __LINE__)

template<typename LayoutA, typename LayoutB, typename LayoutC>
__device__ void run(
	cutlass::gemm::GemmCoord problem_size,
	cutlass::TensorRef<float const, LayoutA> ref_A,
	cutlass::TensorRef<float const, LayoutB> ref_B,
	cutlass::TensorRef<float const, LayoutC> ref_C,
	cutlass::TensorRef<float, LayoutC> ref_D,
	typename EpilogueOutputOp::Params epilogue = EpilogueOutputOp::Params()
	//int split_k_slices=1
) {
	using TGemmKernel = CutlassGemmKernel<LayoutA, LayoutB, LayoutC>;
	ThreadblockSwizzle threadblock_swizzle;
	cutlass::gemm::GemmCoord grid_shape = threadblock_swizzle.get_tiled_shape(
		problem_size, 
		{ThreadblockShape::kM, ThreadblockShape::kN, ThreadblockShape::kK},
		1//split_k_slices
	);
	
	// Initialize the Params structure
	typename TGemmKernel::Params params{
		problem_size,
		grid_shape,
		ref_A.non_const_ref(),
		ref_B.non_const_ref(),
		ref_C.non_const_ref(),
		ref_D,
		epilogue,
		NULL //workspace
	};
	dim3 grid = threadblock_swizzle.get_grid_shape(params.grid_tiled_shape);
	dim3 block(TGemmKernel::kThreadCount, 1, 1);
	int smem_size = int(sizeof(typename TGemmKernel::SharedStorage));
	cutlass::Kernel<TGemmKernel><<<grid, block, smem_size>>>(params);
}

__global__ void gpu_master(float* a, float* b, float* c, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	a += i * n * n;
	b += i * n * n;
	c += i * n * n;
	run<cutlass::layout::RowMajor, cutlass::layout::RowMajor, cutlass::layout::RowMajor>(
		{n,n,n},
		{a, n},
		{b, n},
		{c, n},
		{c, n},
		{1.f, 0.f}
	);
}

int main() {
	float* a = NULL;
	float* b = NULL;
	float* c = NULL;
	int n = 128;
	int m = 7 * 32;
	hipHostMalloc((void**) & a, m * n * n * sizeof(float));
	hipHostMalloc((void**) & b, m * n * n * sizeof(float));
	hipHostMalloc((void**) & c, m * n * n * sizeof(float));
	hipMemset(a, 0x3f, m * n * n * sizeof(float));
	hipMemset(b, 0x3f, m * n * n * sizeof(float));
	hipMemset(c, 0x3f, m * n * n * sizeof(float));
	gpu_master<<<7, 32>>>(a, b, c, n);
	checkCudaErrors(hipPeekAtLastError());
	float* cc = (float*)malloc(m * n * n * sizeof(float));
	hipMemcpy(cc, c, m * n * n * sizeof(float), hipMemcpyDeviceToHost);
	printf("%f %f %f\n", cc[0], cc[n * n + 1], cc[n * n * (m - 1) + n * n - 7]);
	return 0;
}
